
#include <hip/hip_runtime.h>
#include <stdio.h> //for printf
#include <stdint.h> //for uint32_t

/* for now, using uint32_t everywhere, we may overflow. If so, 
   must attempt to use floats for performance reasons. uint64_t is unusably 
   slow. We must use multiplication instead of repeated addition to limit
   error accumulation, but floats should be workable. */
__global__
void multiples(uint32_t step, uint32_t *array, uint32_t n){
    /*this function should compute all multiples of step, and mark 
    indices of prime_array at those points. This function should use
    threadID to divide the problem set */
    uint32_t cpus, id, start, start_mult, end, end_mult;

    cpus = blockDim.x * gridDim.x; //total number of threads

    //ensure extra loop runs if n/cpus has remainder. Can be tuned to ensure
    //at least sqrt(n) thread executions by changing condition
    for( id = blockIdx.x * blockDim.x + threadIdx.x;
            id * (n/cpus) < n;
            id += cpus)
    {

        start = id * (n/cpus); //starts at 0, increments by n/cpus
        if (start < (step*step)){ //ensure we start at n^2, fixes 2, saves work
            start = step*step;
        }
        start_mult = start/step;

        end = (id + 1) * (n/cpus) - 1; //ensure range has no overlap
        if(end > n){ //avoid overflow
            end = n;
        }
        if(start >= end){
            return; //make sure extra threads exit instead of working
        }
        end_mult = end/step;

        printf("thread id %d, num cpus %d, start %d, end %d\n"
               , id, cpus, start, end); //debug

        for(int i = start_mult; i<=end_mult; i++){
            array[step * i] = 1;
        }


    }
}



int main(){
    uint32_t n = 1<<10; //find all primes upto and including this number
    uint32_t *prime_array = (uint32_t *)calloc(n , sizeof(uint32_t)); //allocate and zero
    uint32_t *d_array;
    hipMalloc(&d_array, n * sizeof(uint32_t));
    hipMemcpy(d_array, prime_array, n * sizeof(uint32_t), hipMemcpyHostToDevice);

    for(int loop = 2; loop <= sqrt(n); loop++){ //TODO careful of sqrt here, check for primes only
        multiples<<<1,32>>>(loop,d_array,n);
        hipDeviceSynchronize();
    }

    hipMemcpy(prime_array, d_array, n * sizeof(uint32_t), hipMemcpyDeviceToHost);

    FILE *output_file = fopen("output.txt", "w+");
    
    for(int i=1; i<n; i++){
        if(prime_array[i] == 0){
            fprintf(output_file, "%d ",i);
        }
    }
    fprintf(output_file, "\n");

    fclose(output_file);
    free(prime_array); //clean up
    return 0;
}
